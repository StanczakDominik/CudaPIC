#include "hip/hip_runtime.h"
#include "particles.cuh"

dim3 particleThreads(N_particles_1_axis);
dim3 particleBlocks((N_particles+particleThreads.x - 1)/particleThreads.x);

__device__ int position_to_grid_index(float X, float dx){
    return int(X/dx);
}

__device__ float position_in_cell(float x, float dx){
    int grid_index = position_to_grid_index(x);
    return x - grid_index*dx;
}

__global__ void InitParticleArrays(Particle *d_p, float shiftx, float shifty, float shiftz, float vx, float vy, float vz){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n<N_particles){
        Particle *p = &(d_p[n]);

        int i = n / (int)(N_particles_1_axis*N_particles_1_axis);
        int j = (int) (n/N_particles_1_axis) % N_particles_1_axis;
        int k = n % N_particles_1_axis;
        p->x = L/float(N_particles_1_axis) * i + shiftx;
        p->x = p->x - floor(p->x/L)*L;
        p->y = L/float(N_particles_1_axis) * j + shifty;
        p->y = p->y - floor(p->y/L)*L;
        p->z = L/float(N_particles_1_axis) * k + shiftz;
        p->z = p->z - floor(p->z/L)*L;

        p->vx = vx;
        p->vy = vy;
        p->vz = vz;
    }
}

__device__ int ijk_to_n(int i, int j, int k)
{
    return N_grid * N_grid * (k%N_grid) + N_grid * (j%N_grid) + (i%N_grid);
}

__global__ void scatter_charge(Particle *d_P, float q, float* d_rho){
    int n = blockIdx.x*blockDim.x + threadIdx.x;

    float x = d_P[n].x;
    float y = d_P[n].y;
    float z = d_P[n].z;
    int i = position_to_grid_index(x);
    int j = position_to_grid_index(y);
    int k = position_to_grid_index(z);

    float Xr = position_in_cell(x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell(y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell(z)/dz;
    float Zl = 1 - Zr;

    //TODO: redo this using a reduce
    atomicAdd(&(d_rho[ijk_to_n(i,j,k)]),       q*Xl*Yl*Zl);
    atomicAdd(&(d_rho[ijk_to_n(i+1,j,k)]),     q*Xr*Yl*Zl);
    atomicAdd(&(d_rho[ijk_to_n(i,j+1,k)]),     q*Xl*Yr*Zl);
    atomicAdd(&(d_rho[ijk_to_n(i,j,k+1)]),     q*Xl*Yl*Zr);
    atomicAdd(&(d_rho[ijk_to_n(i+1,j+1,k)]),   q*Xr*Yr*Zl);
    atomicAdd(&(d_rho[ijk_to_n(i+1,j,k+1)]),   q*Xr*Yl*Zr);
    atomicAdd(&(d_rho[ijk_to_n(i,j+1,k+1)]),   q*Xl*Yr*Zr);
    atomicAdd(&(d_rho[ijk_to_n(i+1,j+1,k+1)]), q*Xr*Yr*Zr);
}


__device__ float gather_grid_to_particle(Particle *p, float *grid){
    float x = p->x;
    float y = p->y;
    float z = p->z;
    int i = position_to_grid_index(x);
    int j = position_to_grid_index(y);
    int k = position_to_grid_index(z);

    float Xr = position_in_cell(x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell(y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell(z)/dz;
    float Zl = 1 - Zr;

    float interpolated_scalar = 0.0f;
    interpolated_scalar += grid[ijk_to_n(i,j,k)]      *Xl*Yl*Zl;
    interpolated_scalar += grid[ijk_to_n(i+1,j,k)]    *Xr*Yl*Zl;
    interpolated_scalar += grid[ijk_to_n(i,j+1,k)]    *Xl*Yr*Zl;
    interpolated_scalar += grid[ijk_to_n(i,j,k+1)]    *Xl*Yl*Zr;
    interpolated_scalar += grid[ijk_to_n(i+1,j+1,k)]  *Xr*Yr*Zl;
    interpolated_scalar += grid[ijk_to_n(i+1,j,k+1)]  *Xr*Yl*Zr;
    interpolated_scalar += grid[ijk_to_n(i,j+1,k+1)]  *Xl*Yr*Zr;
    interpolated_scalar += grid[ijk_to_n(i+1,j+1,k+1)]*Xr*Yr*Zr;
    return interpolated_scalar;

}



__global__ void InitialVelocityStep(Particle *d_p, float q, float m, float *d_Ex, float *d_Ey, float *d_Ez){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n<N_particles)
    {
        Particle *p = &(d_p[n]);
        //gather electric field
        float Ex = gather_grid_to_particle(p, d_Ex);
        float Ey = gather_grid_to_particle(p, d_Ey);
        float Ez = gather_grid_to_particle(p, d_Ez);

       //use electric field to accelerate particles
       p->vx -= 0.5f*dt*q/m*Ex;
       p->vy -= 0.5f*dt*q/m*Ey;
       p->vz -= 0.5f*dt*q/m*Ez;
    }
}

__global__ void ParticleKernel(Particle *d_p, float q, float m, float *d_Ex, float *d_Ey, float *d_Ez){
   int n = blockDim.x * blockIdx.x + threadIdx.x;
   if(n<N_particles)
   {
       Particle *p = &(d_p[n]);
       //push positions, enforce periodic boundary conditions

       p->x = p->x + p->vx*dt;
       p->x = p->x - floor(p->x/L)*L;

       p->y = p->y + p->vy*dt;
       p->y = p->y - floor(p->y/L)*L;

       p->z = p->z + p->vz*dt;
       p->z = p->z - floor(p->z/L)*L;

       //gather electric field
       float Ex = gather_grid_to_particle(p, d_Ex);
       float Ey = gather_grid_to_particle(p, d_Ey);
       float Ez = gather_grid_to_particle(p, d_Ez);

       //use electric field to accelerate particles
       p->vx += dt*q/m*Ex;
       p->vy += dt*q/m*Ey;
       p->vz += dt*q/m*Ez;
   }
}


void init_species(Species *s, float shiftx, float shifty, float shiftz, float vx, float vy, float vz){
    s->particles = new Particle[N_particles];
    CUDA_ERROR(hipMalloc((void**)&(s->d_particles), sizeof(Particle)*N_particles));
    printf("initializing particles\n");
    InitParticleArrays<<<particleBlocks, particleThreads>>>(s->d_particles, shiftx, shifty, shiftz, vx, vy, vz);
}

void dump_position_data(Species *s, char* name){
    // printf("Copying particles from GPU to device\n");
    CUDA_ERROR(hipMemcpy(s->particles, s->d_particles, sizeof(Particle)*N_particles, hipMemcpyDeviceToHost));
    // printf("Copied particles from GPU to device\n");
    FILE *initial_position_data = fopen(name, "w");
    for (int i =0; i<N_particles; i += 51)
    {
        Particle *p = &(s->particles[i]);
        fprintf(initial_position_data, "%f %f %f %f %f %f\n", p->x, p->y, p->z, p->vx, p->vy, p->vz);
    }
    // free(s->particles);
    fclose(initial_position_data);
}

// __global__ void diagnostic_reduction_kernel(Species s)
// {
//     int n = blockDim.x * blockIdx.x + threadIdx.x;
//     if(n<N_particles)
//     {
//         Particle *p = &(s.d_particles[n]);
//         float rx = p->x;
//         float ry = p->y;
//         float rz = p->z;
//         float vx = p->vx;
//         float vy = p->vy;
//         float vz = p->vz;
//         float v2 = vx*vx + vy*vy + vz*vz;
//         float vabs = sqrt(v2);
//         //TODO:
//         //  particle field energy requires rewrite
//         //  to keep interpolated field as variable in particle
//         //  rel. easy
//
//         //reduce above variables
//     }
//     if(n == 0)
//     {
//         //s.total_values = reduced variables
//     }
// }
//
// void diagnostics(Species *s)
// {
//     /*
//     calculates:
//     mean velocity
//     mean square of velocity
//     variance as mean square of velocity - mean velocity squared
//
//     kinetic energy
//     potential energy of particles
//     field energy
//     for p in particles:
//
//     warte uśrednienia są:
//     vx, vy, vz, |v|, v^2
//
//     TODO: uśrednić na siatkę!!!!!! wtedy widać jak to ewoluuje!
//
//     1. loop po cząstkach
//         * vx
//         * vy
//         * vz
//         * v^2 = vx^2 + vy^2 + vz^2
//         * |v| = sqrt(v^2)
//         * V(r) (później)
//     2. reduce all powyższe (mozna inplace)
//     3. analiza danych:
//         * podzielić przez N_particles, średnie wielkości
//         * energia kinetyczna: 0.5 m sum v^2
//         * energia potencjalna: 0.5 q sum V(r)
//         * temperatura: 0.5 m (<v^2> - <v>^2)
//     */
//
//     diagnostic_reduction_kernel<<<particleBlocks, particleThreads>>>(s);
//     // float total_kinetic_energy = 0.5f * s.m * s.total_v2;
//     // float avg_modV = s.total_vabs / s.N;
//     // float avg_v2 = s.total_v2 / s.N;
//     // float temperature = 0.5f * s.m * (avg_v2 - avg_modV * avg_modV);
// }
