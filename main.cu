#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define N_particles_1_axis 128 //should be maybe connected to threadsPerBlock somehow
#define N_particles  (N_particles_1_axis*N_particles_1_axis*N_particles_1_axis) //does this compile with const? //2^4^3 = 2^7 = 128
#define L 1.f
#
#define dt 0.01f
#define NT 500
#define N_grid 16
#define N_grid_all (N_grid *N_grid * N_grid)
#define dx (L/float(N_grid))
#define dy dx
#define dz dx
#define epsilon_zero 1.0f
#include <hipfft/hipfft.h>
size_t particle_array_size = N_particles*sizeof(float);
// size_t grid_array_size = N_grid*sizeof(float);

/*
Assumptions:
q=1
m=1
L = 1
*/


dim3 particleThreads(64);
dim3 particleBlocks(N_particles/particleThreads.x);
dim3 gridThreads(16,16,16);
dim3 gridBlocks(N_grid/gridThreads.x, N_grid/gridThreads.y, N_grid/gridThreads.z);

struct Grid
{
    // int N_grid;
    // int N_grid_all;
    // float dx;

    float *rho;
    float *Ex;
    float *Ey;
    float *Ez;

    float *d_rho;
    float *d_Ex;
    float *d_Ey;
    float *d_Ez;
    //fourier transformed versions of grid quantities, for fields solver
    hipfftComplex *d_fourier_rho;
    hipfftComplex *d_fourier_Ex;
    hipfftComplex *d_fourier_Ey;
    hipfftComplex *d_fourier_Ez;

    hipfftHandle plan;

    float *kv;
    float *d_kv; //wave vector for field solver
};

struct Particle
{
    float x;
    float y;
    float z;
    float vx;
    float vy;
    float vz;
};

struct Species
{
    float m;
    float q;
    long int N;

    Particle *particles;
    Particle *d_particles;
};

__global__ void solve_poisson(Grid g){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;
    if(i<N_grid && j<N_grid && k<N_grid)
    {
        float k2 = g.d_kv[i]*g.d_kv[i] + g.d_kv[j]*g.d_kv[j] + g.d_kv[k]*g.d_kv[k];
        if (i==0 && j==0 && k ==0)
        {
            k2 = 1.0f;
        }

        //see: birdsall langdon page 19
        g.d_fourier_Ex[index].x = -g.d_kv[i]*g.d_fourier_rho[index].x/k2/epsilon_zero;
        g.d_fourier_Ex[index].y = -g.d_kv[i]*g.d_fourier_rho[index].y/k2/epsilon_zero;

        g.d_fourier_Ey[index].x = -g.d_kv[j]*g.d_fourier_rho[index].x/k2/epsilon_zero;
        g.d_fourier_Ey[index].y = -g.d_kv[j]*g.d_fourier_rho[index].y/k2/epsilon_zero;

        g.d_fourier_Ez[index].x = -g.d_kv[k]*g.d_fourier_rho[index].x/k2/epsilon_zero;
        g.d_fourier_Ez[index].y = -g.d_kv[k]*g.d_fourier_rho[index].y/k2/epsilon_zero;
    }
}

__global__ void real2complex(float *input, hipfftComplex *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid)
    {
        output[index].x = input[index];
        output[index].y = 0.0f;
    }
}
__global__ void complex2real(hipfftComplex *input, float *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid)
    {
        output[index] = input[index].x/float(N_grid_all);
    }
}

__global__ void scale_down_after_fft(Grid g){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid)
    {
        g.d_Ex[index] /= float(N_grid_all);
        g.d_Ey[index] /= float(N_grid_all);
        g.d_Ez[index] /= float(N_grid_all);
    }
}

__device__ int position_to_grid_index(float X){
    return int(X/dx);
}
__device__ float position_in_cell(float x){
    int grid_index = position_to_grid_index(x);
    return x - grid_index*dx;
}

__global__ void scatter_charge(Species s, Grid g){
    int n = blockIdx.x*blockDim.x + threadIdx.x;

    int i = position_to_grid_index(s.d_particles[n].x);
    int j = position_to_grid_index(s.d_particles[n].y);
    int k = position_to_grid_index(s.d_particles[n].z);

    float Xr = position_in_cell(s.d_particles[n].x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell(s.d_particles[n].y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell(s.d_particles[n].z)/dz;
    float Zl = 1 - Zr;

    //this part is literally hitler - not just unreadable but slow af
    //TODO: redo this using a reduce, maybe?
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]), s.q*Xl*Yl*Zl);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]), s.q*Xr*Yl*Zl);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]), s.q*Xl*Yr*Zl);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]), s.q*Xl*Yl*Zr);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]), s.q*Xr*Yr*Zl);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]), s.q*Xr*Yl*Zr);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]), s.q*Xl*Yr*Zr);
    atomicAdd(&(g.d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]), s.q*Xr*Yr*Zr);
}
__device__ float gather_grid_to_particle(Particle *p, float *grid){
    int i = position_to_grid_index((*p).x);
    int j = position_to_grid_index((*p).y);
    int k = position_to_grid_index((*p).z);

    float Xr = position_in_cell((*p).x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell((*p).y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell((*p).z)/dz;
    float Zl = 1 - Zr;

    float interpolated_scalar = 0.0f;
    //this part is also hitler but not as much
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]*Xl*Yl*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]*Xr*Yl*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]*Xl*Yr*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]*Xl*Yl*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]*Xr*Yr*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]*Xr*Yl*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]*Xl*Yr*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]*Xr*Yr*Zr;
    return interpolated_scalar;

}


__global__ void InitParticleArrays(Species s){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n<N_particles)
    {
        Particle *p = &(s.particles[n]);
        (*p).x = L/float(N_particles_1_axis)*(n%N_particles_1_axis);
        (*p).y = L/float(N_particles_1_axis)*(n/N_particles_1_axis)/float(N_particles_1_axis);
        (*p).z = L/float(N_particles_1_axis)*(n/N_particles_1_axis/N_particles_1_axis);
        (*p).vx = 0.0f;
        (*p).vy = 0.0f;
        (*p).vz = 0.0f;
    }
}
__global__ void InitialVelocityStep(Species s, Grid g){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    {
        Particle *p = &(s.particles[n]);
        //gather electric field
        float Ex = gather_grid_to_particle(*p, g.d_Ex);
        float Ey = gather_grid_to_particle(*p, g.d_Ey);
        float Ez = gather_grid_to_particle(*p, g.d_Ez);

       //use electric field to accelerate particles
       (*p).vx -= 0.5f*dt*qm*Ex;
       (*p).vy -= 0.5f*dt*qm*Ey;
       (*p).vz -= 0.5f*dt*qm*Ez;
    }
}

__global__ void ParticleKernel(Species s, Grid g){
   int n = blockDim.x * blockIdx.x + threadIdx.x;
   if(n<N_particles)
   {
       Particle *p = &(s.particles[n]);
       //push positions, enforce periodic boundary conditions
       (*p).x = fmod(((*p).x + (*p).vx*dt),L);
       (*p).x = fmod(((*p).y + (*p).vy*dt),L);
       (*p).x = fmod(((*p).z + (*p).vz*dt),L);
       //gather electric field
       float Ex = gather_grid_to_particle(X[n], Y[n], Z[n], g.d_Ex);
       float Ey = gather_grid_to_particle(X[n], Y[n], Z[n], g.d_Ey);
       float Ez = gather_grid_to_particle(X[n], Y[n], Z[n], g.d_Ez);

       //use electric field to accelerate particles
       (*p).vx[n] += dt*qm*Ex;
       (*p).vy[n] += dt*qm*Ey;
       (*p).vz[n] += dt*qm*Ez;
   }
}

void init_field_solver(Grid g)
{

    float *k = new float[N_grid];
    for (int i =0; i<=N/2; i++)
    {
        k[i] = i*2*M_PI;
    }
    for (int i = N/2 + 1; i < N; i++)
    {
        k[i] = (i-N)*2*M_PI;
    }


    hipMalloc((void**)&d_kv, sizeof(float)*N_grid);
    hipMemcpy(d_kv, k, sizeof(float)*N_grid, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_fourier_charge, sizeof(hipfftComplex)*N_grid_all);
    hipMalloc((void**)&d_fourier_Ex, sizeof(hipfftComplex)*N_grid_all);
    hipMalloc((void**)&d_fourier_Ey, sizeof(hipfftComplex)*N_grid_all);
    hipMalloc((void**)&d_fourier_Ez, sizeof(hipfftComplex)*N_grid_all);
    hipMalloc((void**)&d_charge, sizeof(float)*N_grid_all);
    hipMalloc((void**)&d_Ex, sizeof(float)*N_grid_all);
    hipMalloc((void**)&d_Ey, sizeof(float)*N_grid_all);
    hipMalloc((void**)&d_Ez, sizeof(float)*N_grid_all);
    hipfftPlan3d(&plan, N_grid, N_grid, N_grid, HIPFFT_R2C);
}
void field_solver(float *d_charge, float *d_Ex, float *d_Ey, float *d_Ez, float *d_k)
{
    hipfftExecR2C(plan, d_charge, d_fourier_charge, HIPFFT_FORWARD);

    solve_poisson<<<gridBlocks, gridThreads>>>(d_fourier_charge,
            d_fourier_Ex, d_fourier_Ey, d_fourier_Ez,d_kv);
    hipfftExecC2R(plan, d_fourier_Ex, d_Ex, HIPFFT_BACKWARD);
    hipfftExecC2R(plan, d_fourier_Ey, d_Ey, HIPFFT_BACKWARD);
    hipfftExecC2R(plan, d_fourier_Ez, d_Ez, HIPFFT_BACKWARD);

    scale_down_after_fft<<<gridBlocks, gridThreads>>>(d_Ex, d_Ey, d_Ez);
}



int main(void){

    hipfftComplex *d_fourier_Ex, *d_fourier_Ey, *d_fourier_Ez,
        *d_fourier_charge;
    hipfftHandle plan;
    float *charge = new float[N_grid_all];
    float *d_charge;
    //TODO: routine checks for cuda status

    float *d_X, *d_Y, *d_Z;
    float *d_Vx, *d_Vy, *d_Vz;

    float *X = new float[N_particles];
    float *Y = new float[N_particles];
    float *Z = new float[N_particles];
    float *Vx = new float[N_particles];
    float *Vy = new float[N_particles];
    float *Vz = new float[N_particles];


    hipMalloc((void**)&d_X, sizeof(float)*N_particles);
    hipMalloc((void**)&d_Y, sizeof(float)*N_particles);
    hipMalloc((void**)&d_Z, sizeof(float)*N_particles);
    hipMalloc((void**)&d_Vx, sizeof(float)*N_particles);
    hipMalloc((void**)&d_Vy, sizeof(float)*N_particles);
    hipMalloc((void**)&d_Vz, sizeof(float)*N_particles);
    InitParticleArrays<<<particleBlocks, particleThreads>>>(d_X, d_Y, d_Z, d_Vx, d_Vy, d_Vz);
    InitialVelocityStep<<<particleBlocks, particleThreads>>>(d_X, d_Y, d_Z, d_Vx, d_Vy, d_Vz);


    hipMalloc((void**)&d_charge, sizeof(float)*N_grid_all);
    hipMemset(&d_charge, sizeof(float)*N_grid_all, 0);

    scatter_charge<<<particleBlocks, particleThreads>>>(d_X, d_Y, d_Z, d_charge, 1);


    hipMemcpy(charge, d_charge, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost);
    FILE *density_data = fopen("init_density.dat", "w");
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f\n", charge[n]);
    }


    hipMemcpy(X, d_X, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
    hipMemcpy(Y, d_Y, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
    hipMemcpy(Z, d_Z, sizeof(float)*N_particles, hipMemcpyDeviceToHost);

    FILE *initial_position_data = fopen("init_position.dat", "w");
    for (int p =0; p<N_particles; p++)
    {
        fprintf(initial_position_data, "%f %f %f\n", X[p], Y[p], Z[p]);
    }

    FILE *trajectory_data = fopen("trajectory.dat", "w");
    for(int i =0; i<NT; i++)
    {
        ParticleKernel<<<particleBlocks, particleThreads>>>(d_X, d_Y, d_Z, d_Vx, d_Vy, d_Vz);
        hipMemcpy(X, d_X, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
        hipMemcpy(Y, d_Y, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
        hipMemcpy(Z, d_Z, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
        for (int p =0; p<10; p++)
        {
            fprintf(trajectory_data,"%f %f %f ", X[p], Y[p], Z[p]);
        }
        fprintf(trajectory_data, "\n");
    }



    hipMemcpy(X, d_X, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
    hipMemcpy(Y, d_Y, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
    hipMemcpy(Z, d_Z, sizeof(float)*N_particles, hipMemcpyDeviceToHost);
    FILE *final_position_data = fopen("final_position.dat", "w");
    for (int p =0; p<N_particles; p++)
    {
        fprintf(final_position_data, "%f %f %f\n", X[p], Y[p], Z[p]);
    }

    hipMemset(&d_charge, sizeof(float)*N_grid_all, 0);
    scatter_charge<<<particleBlocks, particleThreads>>>(d_X, d_Y, d_Z, d_charge, 1);
    hipMemcpy(charge, d_charge, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost);
    FILE *final_density_data = fopen("final_density.dat", "w");
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(final_density_data, "%f\n", charge[n]);
    }

    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);
    hipFree(d_Vx);
    hipFree(d_Vy);
    hipFree(d_Vz);
    hipFree(d_charge);
}
