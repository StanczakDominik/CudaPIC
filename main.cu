#include <stdio.h>
#include <iostream>
#include "grid.cuh"
#include "helpers.cuh"
#include "particles.cuh"
using namespace std;

#define SNAP_EVERY 10
#define NT 1000
#define dt 0.01f

#define N_particles_1_axis 128
#define N_particles  (N_particles_1_axis*N_particles_1_axis*N_particles_1_axis)


void init_timestep(Grid *g, Species *electrons,  Species *ions){
    reset_rho(g);
    scatter_charge(electrons, g);
    scatter_charge(ions, g);

    // debug_field_solver_uniform(g);
    field_solver(g);

    InitialVelocityStep(electrons, g, dt);
    InitialVelocityStep(ions, g, dt);
}


void timestep(Grid *g, Species *electrons,  Species *ions){
	//1. move particles, gather electric fields at their locations, accelerate particles
	SpeciesPush(electrons, g, dt);
	SpeciesPush(ions, g, dt);
	//potential TODO: sort particles?????
    //2. clear charge density for scattering fields to particles charge
    reset_rho(g);

    //3. gather charge from new particle position to grid
    scatter_charge(electrons, g);
    scatter_charge(ions, g);

    //4. use charge density to calculate field
    // debug_field_solver_uniform(g);
    field_solver(g);
}

int main(void){

    int N_grid = 32;

    hipEvent_t startLoop, endLoop;
    hipEventCreate(&startLoop);
    hipEventCreate(&endLoop);


    Grid g;
    init_grid(&g, N_grid);
    CUDA_ERROR(hipDeviceSynchronize());

    Species electrons;
    electrons.q = -ELECTRON_CHARGE;
    electrons.m = ELECTRON_MASS;
    electrons.N = N_particles;
    init_species(&electrons, g.dx*0.06f, g.dx*0.001f, g.dx*0.001f, 1, 0, 0, N_particles_1_axis, g.N_grid, g.dx);
    Species ions;
    // ions.q = ELECTRON_CHARGE;
    // ions.m = PROTON_MASS;
    ions.q = -ELECTRON_CHARGE;
    ions.m = ELECTRON_MASS;
    ions.N = N_particles;
    init_species(&ions, g.dx*0.05f, g.dx*0.001f, g.dx*0.001f, -1, 0, 0, N_particles_1_axis, g.N_grid, g.dx);

    char filename[50];
    sprintf(filename, "data/ions_positions_%d.dat", -1);
    dump_position_data(&ions, filename);
    sprintf(filename, "data/electrons_positions_%d.dat", -1);
    dump_position_data(&electrons, filename);

    init_timestep(&g, &electrons, &ions);

    printf("entering time loop\n");
    hipEventSynchronize(startLoop);
    hipEventRecord(startLoop);
    FILE *energy_data = fopen("energies.dat", "w");
    for(int i =0; i<=NT; i++){
        if (i % SNAP_EVERY == 0)
        {
            printf("Iteration %6d\n", i);
            sprintf(filename, "data/running_density_%d.dat", i);
            dump_density_data(&g, (char*)filename);
            sprintf(filename, "data/ions_positions_%d.dat", i);
            dump_position_data(&ions, filename);
            sprintf(filename, "data/electrons_positions_%d.dat", i);
            dump_position_data(&electrons, filename);
            fprintf(energy_data, "%d ", i);
            fprintf(energy_data, "%f %f ", g.rho_total, g.E_total);
            fprintf(energy_data, "%f %f ", electrons.KE, ions.KE);
            fprintf(energy_data, "%f %f ", electrons.Px, ions.Px);
            fprintf(energy_data, "%f %f ", electrons.Py, ions.Py);
            fprintf(energy_data, "%f %f ", electrons.Pz, ions.Pz);
            fprintf(energy_data, "%f\n", g.E_total + electrons.KE + ions.KE);
        }
        timestep(&g, &electrons, &ions);
    }
    fclose(energy_data);
    hipDeviceSynchronize();
    hipEventSynchronize(endLoop);
    hipEventRecord(endLoop);
    printf("\nfinished time loop\n");
    float loopRuntimeMS = 0;
    hipEventElapsedTime(&loopRuntimeMS, startLoop, endLoop);


    grid_cleanup(&g);
    particle_cleanup(&electrons);
    particle_cleanup(&ions);
}
