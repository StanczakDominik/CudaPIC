
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <iostream>
using namespace std;

#define ELECTRON_MASS 9.10938356e-31
#define PROTON_MASS 1.6726219e-27
#define ELECTRON_CHARGE 1.6021765e-19
#define EPSILON_ZERO 8.854e-12

#define N_particles_1_axis 64 //should be maybe connected to threadsPerBlock somehow
#define N_particles  (N_particles_1_axis*N_particles_1_axis*N_particles_1_axis) //does this compile with const? //2^4^3 = 2^7 = 128
#define L 1e-4
#define dt 1e-11
//TODO: THIS MAY HAVE TO BE CORRECTED
#define NT 1
#define N_grid 16
#define N_grid_all (N_grid *N_grid * N_grid)
#define dx (L/float(N_grid))
#define dy dx
#define dz dx
size_t particle_array_size = N_particles*sizeof(float);
// size_t grid_array_size = N_grid*sizeof(float);

/*
Assumptions:
q=1
m=1
L = 1
*/


dim3 particleThreads(32);
dim3 particleBlocks(N_particles/particleThreads.x);
dim3 gridThreads(16,16,16);
dim3 gridBlocks(N_grid/gridThreads.x, N_grid/gridThreads.y, N_grid/gridThreads.z);
static void CUDA_ERROR( hipError_t err)
{
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s, exiting\n", hipGetErrorString(err));
        exit(-1);
    }
}
struct Grid{
    // int N_grid;
    // int N_grid_all;
    // float dx;

    float *rho;
    float *Ex;
    float *Ey;
    float *Ez;

    float *d_rho;
    float *d_Ex;
    float *d_Ey;
    float *d_Ez;
    //fourier transformed versions of grid quantities, for fields solver
    hipfftComplex *d_fourier_rho;
    hipfftComplex *d_fourier_Ex;
    hipfftComplex *d_fourier_Ey;
    hipfftComplex *d_fourier_Ez;

    hipfftHandle plan_forward;
    hipfftHandle plan_backward;

    float *kv;
    float *d_kv; //wave vector for field solver
};

struct Particle{
    float x;
    float y;
    float z;
    float vx;
    float vy;
    float vz;
};

struct Species{
    float m;
    float q;
    long int N;

    Particle *particles;
    Particle *d_particles;
};

__global__ void solve_poisson(float *d_kv, hipfftComplex *d_fourier_rho, hipfftComplex *d_fourier_Ex, hipfftComplex *d_fourier_Ey, hipfftComplex *d_fourier_Ez){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;
    if(i<N_grid && j<N_grid && k<N_grid){
        float k2 = d_kv[i]*d_kv[i] + d_kv[j]*d_kv[j] + d_kv[k]*d_kv[k];
        if (i==0 && j==0 && k ==0)    {
            k2 = 1.0f;
        }

        //see: birdsall langdon page 19
        d_fourier_Ex[index].x = -d_kv[i]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ex[index].y = -d_kv[i]*d_fourier_rho[index].y/k2/EPSILON_ZERO;

        d_fourier_Ey[index].x = -d_kv[j]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ey[index].y = -d_kv[j]*d_fourier_rho[index].y/k2/EPSILON_ZERO;

        d_fourier_Ez[index].x = -d_kv[k]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ez[index].y = -d_kv[k]*d_fourier_rho[index].y/k2/EPSILON_ZERO;
    }
}

__global__ void real2complex(float *input, hipfftComplex *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid)    {
        output[index].x = input[index];
        output[index].y = 0.0f;
    }
}
__global__ void complex2real(hipfftComplex *input, float *output){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid){
        output[index] = input[index].x/float(N_grid_all);
    }
}

__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid){
        d_Ex[index] /= float(N_grid_all);
        d_Ey[index] /= float(N_grid_all);
        d_Ez[index] /= float(N_grid_all);
    }
}

__global__ void set_grid_array_to_value(float *arr, float value){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i*N_grid;

    if(i<N_grid && j<N_grid && k<N_grid){
        arr[index] = value;
    }
}

void init_grid(Grid *g){
    g->rho = new float[N_grid_all];
    g->Ex = new float[N_grid_all];
    g->Ey = new float[N_grid_all];
    g->Ez = new float[N_grid_all];

    g->kv = new float[N_grid];
    for (int i =0; i<=N_grid/2; i++)
    {
        g->kv[i] = i*2*M_PI;
    }
    for (int i = N_grid/2 + 1; i < N_grid; i++)
    {
        g->kv[i] = (i-N_grid)*2*M_PI;
    }


    CUDA_ERROR(hipMalloc((void**)&(g->d_kv), sizeof(float)*N_grid));
    CUDA_ERROR(hipMemcpy(g->d_kv, g->kv, sizeof(float)*N_grid, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_rho), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ex), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ey), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ez), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_rho), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_rho, g->rho, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ex), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ex, g->Ex, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ey), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ey, g->Ey, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ez), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ez, g->Ez, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    hipfftPlan3d(&(g->plan_forward), N_grid, N_grid, N_grid, HIPFFT_R2C);
    hipfftPlan3d(&(g->plan_backward), N_grid, N_grid, N_grid, HIPFFT_C2R);
}


void debug_field_solver_uniform(Grid *g){
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = k*N_grid*N_grid + j*N_grid + i;
                linear_field_x[index] = 1;
                linear_field_y[index] = 0;
                linear_field_z[index] = 0;
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
void debug_field_solver_linear(Grid *g)
{
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = k*N_grid*N_grid + j*N_grid + i;
                linear_field_x[index] = dx*i;
                linear_field_y[index] = dx*j;
                linear_field_z[index] = dx*k;
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
void debug_field_solver_quadratic(Grid *g)
{
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = k*N_grid*N_grid + j*N_grid + i;
                linear_field_x[index] = (dx*i)*(dx*i);
                linear_field_y[index] = (dx*j)*(dx*j);
                linear_field_z[index] = (dx*k)*(dx*k);
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}

void field_solver(Grid *g){
    hipfftExecR2C(g->plan_forward, g->d_rho, g->d_fourier_rho);

    solve_poisson<<<gridBlocks, gridThreads>>>(g->d_kv, g->d_fourier_rho, g->d_fourier_Ex, g->d_fourier_Ey, g->d_fourier_Ez);
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ex, g->d_Ex);
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ey, g->d_Ey);
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ez, g->d_Ez);

    scale_down_after_fft<<<gridBlocks, gridThreads>>>(g->d_Ex, g->d_Ey, g->d_Ez);
}

__device__ int position_to_grid_index(float X){
    return int(X/dx);
}
__device__ float position_in_cell(float x){
    int grid_index = position_to_grid_index(x);
    return x - grid_index*dx;
}

__global__ void scatter_charge(Particle *d_P, float q, float* d_rho){
    int n = blockIdx.x*blockDim.x + threadIdx.x;

    float x = d_P[n].x;
    float y = d_P[n].y;
    float z = d_P[n].z;
    int i = position_to_grid_index(x);
    int j = position_to_grid_index(y);
    int k = position_to_grid_index(z);

    float Xr = position_in_cell(x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell(y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell(z)/dz;
    float Zl = 1 - Zr;

    //this part is literally hitler - not just unreadable but slow af
    //TODO: redo this using a reduce, maybe?
    atomicAdd(&(d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]), q*Xl*Yl*Zl);
    atomicAdd(&(d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]), q*Xr*Yl*Zl);
    atomicAdd(&(d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]), q*Xl*Yr*Zl);
    atomicAdd(&(d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]), q*Xl*Yl*Zr);
    atomicAdd(&(d_rho[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]), q*Xr*Yr*Zl);
    atomicAdd(&(d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]), q*Xr*Yl*Zr);
    atomicAdd(&(d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]), q*Xl*Yr*Zr);
    atomicAdd(&(d_rho[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]), q*Xr*Yr*Zr);
}
__device__ float gather_grid_to_particle(Particle *p, float *grid){
    float x = p->x;
    float y = p->y;
    float z = p->z;
    int i = position_to_grid_index(x);
    int j = position_to_grid_index(y);
    int k = position_to_grid_index(z);

    float Xr = position_in_cell(x)/dx;
    float Xl = 1 - Xr;
    float Yr = position_in_cell(y)/dy;
    float Yl = 1 - Yr;
    float Zr = position_in_cell(z)/dz;
    float Zl = 1 - Zr;

    float interpolated_scalar = 0.0f;
    //this part is also hitler but not as much
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]*Xl*Yl*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]*Xr*Yl*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]*Xl*Yr*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i)%N_grid]*Xl*Yl*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]*Xr*Yr*Zl;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j)%N_grid + (i+1)%N_grid]*Xr*Yl*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i)%N_grid]*Xl*Yr*Zr;
    interpolated_scalar += grid[N_grid * N_grid * (k+1)%N_grid + N_grid * (j+1)%N_grid + (i+1)%N_grid]*Xr*Yr*Zr;
    return interpolated_scalar;

}


__global__ void InitParticleArrays(Particle *d_p, float shiftx, float shifty, float shiftz){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n<N_particles){
        Particle *p = &(d_p[n]);

        int i = n / (int)(N_particles_1_axis*N_particles_1_axis);
        int j = (int) (n/N_particles_1_axis) % N_particles_1_axis;
        int k = n % N_particles_1_axis;
        p->x = L/float(N_particles_1_axis) * i + shiftx;
        p->x = p->x - floor(p->x/L)*L;
        p->y = L/float(N_particles_1_axis) * j + shifty;
        p->y = p->y - floor(p->y/L)*L;
        p->z = L/float(N_particles_1_axis) * k + shiftz;
        p->z = p->z - floor(p->z/L)*L;

        p->vx = 0.0f;
        p->vy = 0.0f;
        p->vz = 0.0f;
    }
}
__global__ void InitialVelocityStep(Particle *d_p, float q, float m, float *d_Ex, float *d_Ey, float *d_Ez){
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    {
        Particle *p = &(d_p[n]);
        //gather electric field
        float Ex = gather_grid_to_particle(p, d_Ex);
        float Ey = gather_grid_to_particle(p, d_Ey);
        float Ez = gather_grid_to_particle(p, d_Ez);

       //use electric field to accelerate particles
       p->vx -= 0.5f*dt*q/m*Ex;
       p->vy -= 0.5f*dt*q/m*Ey;
       p->vz -= 0.5f*dt*q/m*Ez;
    }
}

__global__ void ParticleKernel(Particle *d_p, float q, float m, float *d_Ex, float *d_Ey, float *d_Ez){
   int n = blockDim.x * blockIdx.x + threadIdx.x;
   if(n<N_particles)
   {
       Particle *p = &(d_p[n]);
       //push positions, enforce periodic boundary conditions

       p->x = p->x + p->vx*dt;
       p->x = p->x - floor(p->x/L)*L;

       p->y = p->y + p->vy*dt;
       p->y = p->y - floor(p->y/L)*L;

       p->z = p->z + p->vz*dt;
       p->z = p->z - floor(p->z/L)*L;

       //gather electric field
       float Ex = gather_grid_to_particle(p, d_Ex);
       float Ey = gather_grid_to_particle(p, d_Ey);
       float Ez = gather_grid_to_particle(p, d_Ez);

       //use electric field to accelerate particles
       p->vx += dt*q/m*Ex;
       p->vy += dt*q/m*Ey;
       p->vz += dt*q/m*Ez;
   }
}


void init_species(Species *s, float shiftx, float shifty, float shiftz){
    s->particles = new Particle[N_particles];
    CUDA_ERROR(hipMalloc((void**)&(s->d_particles), sizeof(Particle)*N_particles));
    cout << "initializing particles" << endl;
    InitParticleArrays<<<particleBlocks, particleThreads>>>(s->d_particles, shiftx, shifty, shiftz);
}

void dump_density_data(Grid *g, char* name){
    cout << "dumping" << endl;
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f %f %f %f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
    }
    for(int n = 0; n < 10; n++)
    {
        printf("%f %f %f %f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
    }
    // free(g->rho);
}

void dump_position_data(Species *s, char* name){
    cout << "Copying particles from GPU to device"<< endl;
    CUDA_ERROR(hipMemcpy(s->particles, s->d_particles, sizeof(Particle)*N_particles, hipMemcpyDeviceToHost));
    cout << "Copied particles from GPU to device"<< endl;
    FILE *initial_position_data = fopen(name, "w");
    for (int i =0; i<N_particles; i++)
    {
        Particle *p = &(s->particles[i]);
        fprintf(initial_position_data, "%f %f %f %f %f %f\n", p->x, p->y, p->z, p->vx, p->vy, p->vz);
    }
    // free(s->particles);
}

void init_timestep(Grid *g, Species *electrons,  Species *ions){
    set_grid_array_to_value<<<gridBlocks, gridThreads>>>(g->d_rho, 0);
    CUDA_ERROR(hipDeviceSynchronize());
    scatter_charge<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, g->d_rho);
    scatter_charge<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());

    debug_field_solver_uniform(g);
    // field_solver(g);
    CUDA_ERROR(hipDeviceSynchronize());

    InitialVelocityStep<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, electrons->m, g->d_Ex, g->d_Ey, g->d_Ez);
    InitialVelocityStep<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, ions->m, g->d_Ex, g->d_Ey, g->d_Ez);
    CUDA_ERROR(hipDeviceSynchronize());
}


void timestep(Grid *g, Species *electrons,  Species *ions){
	//1. move particles, gather electric fields at their locations, accelerate particles
	ParticleKernel<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, electrons->m, g->d_Ex, g->d_Ey, g->d_Ez);
	ParticleKernel<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, ions->m, g->d_Ex, g->d_Ey, g->d_Ez);
	//potential TODO: sort particles?????
    //2. clear charge density for scattering fields to particles charge
    set_grid_array_to_value<<<gridBlocks, gridThreads>>>(g->d_rho, 0);
    CUDA_ERROR(hipDeviceSynchronize());


    //3. gather charge from new particle position to grid
    //TODO: note that I may need to cudaSyncThreads between these steps
    scatter_charge<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, g->d_rho);
    scatter_charge<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());

    //4. use charge density to calculate field
    debug_field_solver_uniform(g);
    // field_solver(g);
    CUDA_ERROR(hipDeviceSynchronize());
}

int main(void){
    Grid g;
    init_grid(&g);
    dump_density_data(&g, "initial_density.dat");

    Species electrons;
    electrons.q = -ELECTRON_CHARGE;
    electrons.m = ELECTRON_MASS;
    electrons.N = N_particles;
    init_species(&electrons, L/100.0f, 0, 0);

    Species ions;
    ions.q = +ELECTRON_CHARGE;
    ions.m = PROTON_MASS;
    ions.N = N_particles;
    init_species(&ions, 0, 0, 0);

    init_timestep(&g, &electrons, &ions);

    dump_position_data(&ions, "ions_positions.dat");
    dump_position_data(&electrons, "electrons_positions.dat.");
    dump_density_data(&g, "initial_density.dat");
    cout << "entering time loop" << endl;
    for(int i =0; i<NT; i++){
        timestep(&g, &electrons, &ions);
    }
    cout << "finished time loop" << endl;

    dump_position_data(&ions, "final_ions_positions.dat");
    dump_position_data(&electrons, "final_electrons_positions.dat.");
    dump_density_data(&g, "final_density.dat");
    CUDA_ERROR(hipFree(electrons.d_particles));
    CUDA_ERROR(hipFree(g.d_rho));
    CUDA_ERROR(hipFree(g.d_Ex));
    CUDA_ERROR(hipFree(g.d_Ey));
    CUDA_ERROR(hipFree(g.d_Ez));
    CUDA_ERROR(hipFree(g.d_fourier_Ex));
    CUDA_ERROR(hipFree(g.d_fourier_Ey));
    CUDA_ERROR(hipFree(g.d_fourier_Ez));
    CUDA_ERROR(hipFree(g.d_fourier_rho));
}
