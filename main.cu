#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include "grid.h"
#include "helpers.h"
#include "particles.h"
using namespace std;

void init_timestep(Grid *g, Species *electrons,  Species *ions){
    set_grid_array_to_value<<<gridBlocks, gridThreads>>>(g->d_rho, 0);
    CUDA_ERROR(hipDeviceSynchronize());
    scatter_charge<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());
    scatter_charge<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());

    // debug_field_solver_sine(g);
    field_solver(g);
    CUDA_ERROR(hipDeviceSynchronize());

    InitialVelocityStep<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, electrons->m, g->d_Ex, g->d_Ey, g->d_Ez);
    InitialVelocityStep<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, ions->m, g->d_Ex, g->d_Ey, g->d_Ez);
    CUDA_ERROR(hipDeviceSynchronize());
}


void timestep(Grid *g, Species *electrons,  Species *ions){
	//1. move particles, gather electric fields at their locations, accelerate particles
	ParticleKernel<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, electrons->m, g->d_Ex, g->d_Ey, g->d_Ez);
	ParticleKernel<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, ions->m, g->d_Ex, g->d_Ey, g->d_Ez);
	//potential TODO: sort particles?????
    //2. clear charge density for scattering fields to particles charge
    set_grid_array_to_value<<<gridBlocks, gridThreads>>>(g->d_rho, 0);
    CUDA_ERROR(hipDeviceSynchronize());


    //3. gather charge from new particle position to grid
    //TODO: note that I may need to cudaSyncThreads between these steps
    scatter_charge<<<particleBlocks, particleThreads>>>(electrons->d_particles, electrons->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());
    scatter_charge<<<particleBlocks, particleThreads>>>(ions->d_particles, ions->q, g->d_rho);
    CUDA_ERROR(hipDeviceSynchronize());

    //4. use charge density to calculate field
    field_solver(g);
    CUDA_ERROR(hipDeviceSynchronize());
}

__global__ diagnostic_reduction_kernel(Species *s)
{
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if(n<N_particles)
    {
        Particle *p = &(d_p[n]);
        float rx = p->rx;
        float ry = p->ry;
        float rz = p->rz;
        float vx = p->vx;
        float vy = p->vy;
        float vz = p->vz;
        float v2 = vx*vx + vy*vy + vz*vz;
        float vabs = sqrt(v2);
        //TODO:
        //  particle field energy requires rewrite
        //  to keep interpolated field as variable in particle
        //  rel. easy

        //reduce above variables
    }
    if(n == 0)
    {
        //s.total_values = reduced variables
    }
}

void diagnostics(Species *s)
{
    /*
    calculates:
    mean velocity
    mean square of velocity
    variance as mean square of velocity - mean velocity squared

    kinetic energy
    potential energy of particles
    field energy
    for p in particles:

    warte uśrednienia są:
    vx, vy, vz, |v|, v^2

    TODO: uśrednić na siatkę!!!!!! wtedy widać jak to ewoluuje!

    1. loop po cząstkach
        * vx
        * vy
        * vz
        * v^2 = vx^2 + vy^2 + vz^2
        * |v| = sqrt(v^2)
        * V(r) (później)
    2. reduce all powyższe (mozna inplace)
    3. analiza danych:
        * podzielić przez N_particles, średnie wielkości
        * energia kinetyczna: 0.5 m sum v^2
        * energia potencjalna: 0.5 q sum V(r)
        * temperatura: 0.5 m (<v^2> - <v>^2)
    */

    diagnostic_reduction_kernel<<<particleBlocks, particleThreads>>>(s);
    float total_kinetic_energy = 0.5f * s.m * s.total_v2;
    float avg_modV = s.total_vabs / s.N;
    float avg_v2 = s.total_v2 / s.N;
    float temperature = 0.5f * s.m * (avg_v2 - avg_modV * avg_modV);
}

int main(void){


    hipEvent_t startLoop, endLoop;
    hipEventCreate(&startLoop);
    hipEventCreate(&endLoop);


    Grid g;
    init_grid(&g);

    Species electrons;
    electrons.q = -ELECTRON_CHARGE;
    electrons.m = ELECTRON_MASS;
    electrons.N = N_particles;
    init_species(&electrons, L/100.0f, 0, 0);

    Species ions;
    ions.q = +ELECTRON_CHARGE;
    ions.m = PROTON_MASS;
    ions.N = N_particles;
    init_species(&ions, 0, 0, 0);
    //TODO: initialize for two stream instability
    init_timestep(&g, &electrons, &ions);

    CUDA_ERROR(hipGetLastError());
    // dump_position_data(&ions, "ions_positions.dat");
    // dump_position_data(&electrons, "electrons_positions.dat");
    dump_density_data(&g, "initial_density.dat");

    cout << "entering time loop" << endl;
    hipEventSynchronize(startLoop);
    hipEventRecord(startLoop);
    for(int i =0; i<NT; i++){
        char* filename = new char[100];
        sprintf(filename, "gfx/running_density_%d.dat", i);
        dump_running_density_data(&g, filename);
        timestep(&g, &electrons, &ions);
        printf("Iteration %d\r", i);
    }

    hipDeviceSynchronize();
    hipEventSynchronize(endLoop);
    hipEventRecord(endLoop);
    cout << endl << "finished time loop" << endl;

    float loopRuntimeMS = 0;
    hipEventElapsedTime(&loopRuntimeMS, startLoop, endLoop);

    printf("Particles Threads per block Blocks Runtime\n");
    printf("%8d %17d %6d %f\n", N_particles, particleThreads.x, particleBlocks.x, loopRuntimeMS);
    if (loopRuntimeMS > 0.0001)
    {
        char* filename = new char[100];
        sprintf(filename, "benchmark/pb_%d_%d_%d.bdat", N_particles, particleThreads.x, particleBlocks.x);
        FILE *benchmark = fopen(filename, "w");
        fprintf(benchmark, "Particles Threads per block Blocks\tRuntime\n");
        fprintf(benchmark, "%8d %17d %6d %f\n", N_particles, particleThreads.x, particleBlocks.x, loopRuntimeMS);
        fclose(benchmark);
    }
    else
    {
        printf("Not saved!\n");
    }

    dump_density_data(&g, "final_density.dat");


    CUDA_ERROR(hipFree(electrons.d_particles));
    CUDA_ERROR(hipFree(g.d_rho));
    CUDA_ERROR(hipFree(g.d_Ex));
    CUDA_ERROR(hipFree(g.d_Ey));
    CUDA_ERROR(hipFree(g.d_Ez));
    CUDA_ERROR(hipFree(g.d_fourier_Ex));
    CUDA_ERROR(hipFree(g.d_fourier_Ey));
    CUDA_ERROR(hipFree(g.d_fourier_Ez));
    CUDA_ERROR(hipFree(g.d_fourier_rho));
}
