#include "hip/hip_runtime.h"
#include "grid.cuh"

dim3 gridThreads(N_grid/2,N_grid/2,N_grid/2);
dim3 gridBlocks((N_grid+gridThreads.x-1)/gridThreads.x, (N_grid + gridThreads.y - 1)/gridThreads.y, (N_grid+gridThreads.z-1)/gridThreads.z);

__device__ int position_to_grid_index(float X){
    return int(X/dx);
}
__device__ float position_in_cell(float x){
    int grid_index = position_to_grid_index(x);
    return x - grid_index*dx;
}

__global__ void solve_poisson(float *d_kv, hipfftComplex *d_fourier_rho, hipfftComplex *d_fourier_Ex, hipfftComplex *d_fourier_Ey, hipfftComplex *d_fourier_Ez){
    /*solve poisson equation
    d_kv: wave vector
    d_fourier_rho: complex array of fourier transformed charge densities
    d_fourier_E(i):
    */
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int index = k*N_grid*N_grid + j*N_grid + i;
    if(i<N_grid && j<N_grid && k<N_grid){
	//wave vector magnitude squared
        float k2 = d_kv[i]*d_kv[i] + d_kv[j]*d_kv[j] + d_kv[k]*d_kv[k];
        if (i==0 && j==0 && k ==0)    {
            k2 = 1.0f; //dodge a bullet with a division by zero
        }

        //see: Birdsall Langdon, Plasma Physics via Computer Simulation, page 19
        d_fourier_Ex[index].x = -d_kv[i]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ex[index].y = -d_kv[i]*d_fourier_rho[index].y/k2/EPSILON_ZERO;

        d_fourier_Ey[index].x = -d_kv[j]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ey[index].y = -d_kv[j]*d_fourier_rho[index].y/k2/EPSILON_ZERO;

        d_fourier_Ez[index].x = -d_kv[k]*d_fourier_rho[index].x/k2/EPSILON_ZERO;
        d_fourier_Ez[index].y = -d_kv[k]*d_fourier_rho[index].y/k2/EPSILON_ZERO;
    }
}

__global__ void real2complex(float *input, hipfftComplex *output){
    //converts array of floats to array of real complex numbers
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid)    {
        output[index].x = input[index];
        output[index].y = 0.0f;
    }
}
__global__ void complex2real(hipfftComplex *input, float *output){
    //converts array of complex inputs to floats (discards)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        output[index] = input[index].x/float(N_grid_all);
    }
}

__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        d_Ex[index] /= float(N_grid_all);
        d_Ey[index] /= float(N_grid_all);
        d_Ez[index] /= float(N_grid_all);
    }
}

__global__ void set_grid_array_to_value(float *arr, float value){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if((i<N_grid) && (j<N_grid) && (k<N_grid)){
        arr[index] = value;
    }
}

void init_grid(Grid *g){
    g->rho = new float[N_grid_all];
    g->Ex = new float[N_grid_all];
    g->Ey = new float[N_grid_all];
    g->Ez = new float[N_grid_all];

    g->kv = new float[N_grid];
    for (int i =0; i<=N_grid/2; i++)
    {
        g->kv[i] = i*2*M_PI;
    }
    for (int i = N_grid/2 + 1; i < N_grid; i++)
    {
        g->kv[i] = (i-N_grid)*2*M_PI;
    }


    CUDA_ERROR(hipMalloc((void**)&(g->d_kv), sizeof(float)*N_grid));
    CUDA_ERROR(hipMemcpy(g->d_kv, g->kv, sizeof(float)*N_grid, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_rho), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ex), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ey), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_fourier_Ez), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_rho), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_rho, g->rho, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ex), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ex, g->Ex, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ey), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ey, g->Ey, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ez), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ez, g->Ez, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    hipfftPlan3d(&(g->plan_forward), N_grid, N_grid, N_grid, HIPFFT_R2C);
    hipfftPlan3d(&(g->plan_backward), N_grid, N_grid, N_grid, HIPFFT_C2R);
}


void debug_field_solver_uniform(Grid *g){
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = i*N_grid*N_grid + j*N_grid + k;
                linear_field_x[index] = 1000;
                linear_field_y[index] = 0;
                linear_field_z[index] = 0;
                // printf("%d %f %f %f\n", index, linear_field_x[index], linear_field_y[index],linear_field_z[index]);
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
void debug_field_solver_sine(Grid *g)
{
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = i*N_grid*N_grid + j*N_grid + k;
                linear_field_x[index] = 1000*sin(2*M_PI*((float)k/(float)N_grid));
                linear_field_y[index] = 1000*sin(2*M_PI*((float)j/(float)N_grid));
                linear_field_z[index] = 1000*sin(2*M_PI*((float)i/(float)N_grid));
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
void field_solver(Grid *g){
    hipfftExecR2C(g->plan_forward, g->d_rho, g->d_fourier_rho);
    CUDA_ERROR(hipDeviceSynchronize());
    solve_poisson<<<gridBlocks, gridThreads>>>(g->d_kv, g->d_fourier_rho, g->d_fourier_Ex, g->d_fourier_Ey, g->d_fourier_Ez);
    CUDA_ERROR(hipDeviceSynchronize());
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ex, g->d_Ex);
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ey, g->d_Ey);
    hipfftExecC2R(g->plan_backward, g->d_fourier_Ez, g->d_Ez);

    scale_down_after_fft<<<gridBlocks, gridThreads>>>(g->d_Ex, g->d_Ey, g->d_Ez);
    CUDA_ERROR(hipDeviceSynchronize());
}

void dump_density_data(Grid *g, char* name){
    printf("dumping\n");
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    float rho_total = 0.0f;
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f %.2f %.2f %.2f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
        // printf("%d %f %f %f %f\n", n, g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
        rho_total += g->rho[n];
    }
    printf("rho total: %f\n", rho_total);
}

void dump_running_density_data(Grid *g, char* name){
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f %.0f %.0f %.0f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
    }
    // fclose(density_data);
}
