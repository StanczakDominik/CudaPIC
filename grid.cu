#include "hip/hip_runtime.h"
#include "grid.cuh"


__global__ void solve_poisson(float *d_kv, hipfftComplex *d_F_rho,
        hipfftComplex *d_F_Ex, hipfftComplex *d_F_Ey, hipfftComplex *d_F_Ez,
        int N_grid, int N_grid_all){
    /*solve poisson equation
    d_kv: wave vector
    d_F_rho: complex array of fourier transformed charge densities
    d_F_E(i):
    */
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int index = k*N_grid*N_grid + j*N_grid + i;
    if(i<N_grid && j<N_grid && k<N_grid){
        float k2inverse; //wave vector magnitude squared
        if (i==0 && j==0 && k ==0)    {
            k2inverse = 1.0f; //dodge a bullet with a division by zero
        }
        else
        {
            k2inverse = 1/(d_kv[i]*d_kv[i] + d_kv[j]*d_kv[j] + d_kv[k]*d_kv[k]);
        }

        //see: Birdsall Langdon, Plasma Physics via Computer Simulation, page 19
        d_F_Ex[index].x = -d_kv[i]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ex[index].y = -d_kv[i]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;

        d_F_Ey[index].x = -d_kv[j]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ey[index].y = -d_kv[j]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;

        d_F_Ez[index].x = -d_kv[k]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ez[index].y = -d_kv[k]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;


    }
}

__global__ void real2complex(float *input, hipfftComplex *output, int N_grid){
    //converts array of floats to array of real complex numbers
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid)    {
        output[index].x = input[index];
        output[index].y = 0.0f;
    }
}
__global__ void complex2real(hipfftComplex *input, float *output, int N_grid, int N_grid_all){
    //converts array of complex inputs to floats (discards)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        output[index] = input[index].x/float(N_grid_all);
    }
}

__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez, int N_grid, int N_grid_all){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        d_Ex[index] /= float(N_grid_all);
        d_Ey[index] /= float(N_grid_all);
        d_Ez[index] /= float(N_grid_all);
    }
}

__global__ void set_grid_array_to_value(float *arr, float value, int N_grid){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if((i<N_grid) && (j<N_grid) && (k<N_grid)){
        arr[index] = value;
    }
}


/*
* HIGH LEVEL KERNEL WRAPPERS
*/

void init_grid(Grid *g, int N_grid, int N_grid_all){
    g->rho = new float[N_grid_all];
    g->Ex = new float[N_grid_all];
    g->Ey = new float[N_grid_all];
    g->Ez = new float[N_grid_all];

    g->kv = new float[N_grid];
    for (int i =0; i<=N_grid/2; i++)
    {
        g->kv[i] = i*2*M_PI;
    }
    for (int i = N_grid/2 + 1; i < N_grid; i++)
    {
        g->kv[i] = (i-N_grid)*2*M_PI;
    }


    CUDA_ERROR(hipMalloc((void**)&(g->d_kv), sizeof(float)*N_grid));
    CUDA_ERROR(hipMemcpy(g->d_kv, g->kv, sizeof(float)*N_grid, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_F_rho), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ex), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ey), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ez), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_rho), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_rho, g->rho, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ex), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ex, g->Ex, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ey), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ey, g->Ey, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));
    CUDA_ERROR(hipMalloc((void**)&(g->d_Ez), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ez, g->Ez, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    hipfftPlan3d(&(g->plan_forward), N_grid, N_grid, N_grid, HIPFFT_R2C);
    hipfftPlan3d(&(g->plan_backward), N_grid, N_grid, N_grid, HIPFFT_C2R);
}

void field_solver(Grid *g, int N_grid, int N_grid_all, dim3 gridBlocks, dim3 gridThreads){
    hipfftExecR2C(g->plan_forward, g->d_rho, g->d_F_rho);
    CUDA_ERROR(hipDeviceSynchronize());
    solve_poisson<<<gridBlocks, gridThreads>>>(g->d_kv, g->d_F_rho, g->d_F_Ex, g->d_F_Ey, g->d_F_Ez, N_grid, N_grid_all);
    CUDA_ERROR(hipDeviceSynchronize());
    hipfftExecC2R(g->plan_backward, g->d_F_Ex, g->d_Ex);
    hipfftExecC2R(g->plan_backward, g->d_F_Ey, g->d_Ey);
    hipfftExecC2R(g->plan_backward, g->d_F_Ez, g->d_Ez);

    scale_down_after_fft<<<gridBlocks, gridThreads>>>(g->d_Ex, g->d_Ey, g->d_Ez, N_grid, N_grid_all);
    CUDA_ERROR(hipDeviceSynchronize());
}

void dump_density_data(Grid *g, char* name, int N_grid, int N_grid_all){
    printf("dumping\n");
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    float rho_total = 0.0f;
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f %.2f %.2f %.2f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
        rho_total += g->rho[n];
    }
    printf("rho total: %f\n", rho_total);
}

void dump_running_density_data(Grid *g, char* name, int N_grid, int N_grid_all){
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*N_grid_all, hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    for (int n = 0; n < N_grid_all; n++)
    {
        fprintf(density_data, "%f %.2f %.2f %.2f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
    }
}



/*
*   DEBUG SOLVERS
*
*/
void debug_field_solver_uniform(Grid *g, int N_grid, int N_grid_all){
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = i*N_grid*N_grid + j*N_grid + k;
                linear_field_x[index] = 1000;
                linear_field_y[index] = 0;
                linear_field_z[index] = 0;
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
void debug_field_solver_sine(Grid *g, int N_grid, int N_grid_all)
{
    float* linear_field_x = new float[N_grid_all];
    float* linear_field_y = new float[N_grid_all];
    float* linear_field_z = new float[N_grid_all];
    for(int i = 0; i<N_grid;  i++){
        for(int j = 0; j<N_grid;  j++){
            for(int k = 0; k<N_grid;  k++){
                int index = i*N_grid*N_grid + j*N_grid + k;
                linear_field_x[index] = 1000*sin(2*M_PI*((float)k/(float)N_grid));
                linear_field_y[index] = 1000*sin(2*M_PI*((float)j/(float)N_grid));
                linear_field_z[index] = 1000*sin(2*M_PI*((float)i/(float)N_grid));
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*N_grid_all, hipMemcpyHostToDevice);
}
