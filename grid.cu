#include "hip/hip_runtime.h"
#include "grid.cuh"


__global__ void solve_poisson(float *d_kv, hipfftComplex *d_F_rho,
        hipfftComplex *d_F_Ex, hipfftComplex *d_F_Ey, hipfftComplex *d_F_Ez,
        int N_grid, int N_grid_all){
    /*solve poisson equation
    d_kv: wave vector
    d_F_rho: complex array of fourier transformed charge densities
    d_F_E(i):
    */
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;

    int index = k*N_grid*N_grid + j*N_grid + i;
    if(i<N_grid && j<N_grid && k<N_grid){
        float k2inverse; //wave vector magnitude squared
        if (i==0 && j==0 && k ==0)    {
            k2inverse = 1.0f; //dodge a bullet with a division by zero
        }
        else
        {
            k2inverse = 1.0f/(d_kv[i]*d_kv[i] + d_kv[j]*d_kv[j] + d_kv[k]*d_kv[k]);
        }

        //see: Birdsall Langdon, Plasma Physics via Computer Simulation, page 19
        d_F_Ex[index].x = -d_kv[i]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ex[index].y = -d_kv[i]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;

        d_F_Ey[index].x = -d_kv[j]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ey[index].y = -d_kv[j]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;

        d_F_Ez[index].x = -d_kv[k]*d_F_rho[index].x*k2inverse/EPSILON_ZERO;
        d_F_Ez[index].y = -d_kv[k]*d_F_rho[index].y*k2inverse/EPSILON_ZERO;
    }
}

__global__ void real2complex(float *input, hipfftComplex *output, int N_grid){
    //converts array of floats to array of real complex numbers
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid)    {
        output[index].x = input[index];
        output[index].y = 0.0f;
    }
}
__global__ void complex2real(hipfftComplex *input, float *output, int N_grid, int N_grid_all){
    //converts array of complex inputs to floats (discards)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        output[index] = input[index].x/float(N_grid_all);
    }
}

__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez, int N_grid, int N_grid_all){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if(i<N_grid && j<N_grid && k<N_grid){
        d_Ex[index] /= float(N_grid_all);
        d_Ey[index] /= float(N_grid_all);
        d_Ez[index] /= float(N_grid_all);
    }
}

__global__ void set_grid_array_to_value(float *arr, float value, int N_grid){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    int index = k*N_grid*N_grid + j*N_grid + i;

    if((i<N_grid) && (j<N_grid) && (k<N_grid)){
        arr[index] = value;
    }
}

void reset_rho(Grid *g)
{
    CUDA_ERROR(hipDeviceSynchronize());
    set_grid_array_to_value<<<g->gridBlocks, g->gridThreads>>>(g->d_rho, 0, g->N_grid);
}


/*
* HIGH LEVEL KERNEL WRAPPERS
*/

void init_grid(Grid *g, int N_grid){
    int N_grid_all = N_grid * N_grid * N_grid;
    g->N_grid = N_grid;
    g->N_grid_all = N_grid_all;
    g->rho = new float[N_grid_all];
    g->Ex = new float[N_grid_all];
    g->Ey = new float[N_grid_all];
    g->Ez = new float[N_grid_all];

    g->kv = new float[N_grid];
    for (int i =0; i<=N_grid/2; i++)
    {
        g->kv[i] = i*2*M_PI;
    }
    for (int i = N_grid/2 + 1; i < N_grid; i++)
    {
        g->kv[i] = (i-N_grid)*2*M_PI;
    }
    g->dx = (L/float(N_grid));
    g->dy = g->dx;
    g->dz = g->dx;

    g->sum_results = new float[4];
    CUDA_ERROR(hipMalloc((void**)&(g->d_sum_results), sizeof(float)*4));

    printf("Initializing grid\ndx: %f N_grid: %d N_grid_all: %d\n", g->dx, g->N_grid, g->N_grid_all);

    g->gridThreads = dim3(gThreadsSingle,gThreadsSingle,gThreadsSingle);
    g->gridBlocks = dim3((N_grid+g->gridThreads.x-1)/g->gridThreads.x,
        (N_grid + g->gridThreads.y - 1)/g->gridThreads.y, (N_grid+g->gridThreads.z-1)/g->gridThreads.z);

    // printf("%d %d %d\n", g->gridThreads.x, g->gridThreads.y, g->gridThreads.z);
    // printf("%d %d %d\n", g->gridBlocks.x, g->gridBlocks.y, g->gridBlocks.z);
    CUDA_ERROR(hipMalloc((void**)&(g->d_kv), sizeof(float)*N_grid));
    CUDA_ERROR(hipMemcpy(g->d_kv, g->kv, sizeof(float)*N_grid, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_F_rho), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ex), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ey), sizeof(hipfftComplex)*N_grid_all));
    CUDA_ERROR(hipMalloc((void**)&(g->d_F_Ez), sizeof(hipfftComplex)*N_grid_all));


    CUDA_ERROR(hipMalloc((void**)&(g->d_rho), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_rho, g->rho, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_Ex), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ex, g->Ex, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_Ey), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ey, g->Ey, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_Ez), sizeof(float)*N_grid_all));
    CUDA_ERROR(hipMemcpy(g->d_Ez, g->Ez, sizeof(float)*N_grid_all, hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&(g->d_Rrho), sizeof(float)*(N_grid_all+gThreadsAll-1)/gThreadsAll));
    CUDA_ERROR(hipMalloc((void**)&(g->d_REx), sizeof(float)*(N_grid_all+gThreadsAll-1)/gThreadsAll));
    CUDA_ERROR(hipMalloc((void**)&(g->d_REy), sizeof(float)*(N_grid_all+gThreadsAll-1)/gThreadsAll));
    CUDA_ERROR(hipMalloc((void**)&(g->d_REz), sizeof(float)*(N_grid_all+gThreadsAll-1)/gThreadsAll));

    hipfftPlan3d(&(g->plan_forward), N_grid, N_grid, N_grid, HIPFFT_R2C);
    hipfftPlan3d(&(g->plan_backward), N_grid, N_grid, N_grid, HIPFFT_C2R);
}

__global__ void reduce_fields(float *d_rho, float *d_Ex, float* d_Ey, float* d_Ez, float *d_Rrho, float* d_REx, float* d_REy, float* d_REz, int N)
{
    __shared__ float rho_array[gThreadsAll];
    __shared__ float Ex_array[gThreadsAll];
    __shared__ float Ey_array[gThreadsAll];
    __shared__ float Ez_array[gThreadsAll];
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N){
        for (int s = blockDim.x / 2; s > 0; s >>= 1){
            if ( threadIdx.x < s)
            {
                rho_array[threadIdx.x] += d_rho[threadIdx.x + s];
                Ex_array[threadIdx.x] += d_Ex[threadIdx.x + s] * d_Ex[threadIdx.x + s];
                Ey_array[threadIdx.x] += d_Ey[threadIdx.x + s] * d_Ey[threadIdx.x + s];
                Ez_array[threadIdx.x] += d_Ez[threadIdx.x + s] * d_Ez[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x ==0){
            d_Rrho[blockIdx.x] = rho_array[0];
            d_REx[blockIdx.x] = Ex_array[0];
            d_REy[blockIdx.x] = Ey_array[0];
            d_REz[blockIdx.x] = Ez_array[0];
        }
    }
}


void field_solver(Grid *g){
    CUDA_ERROR(hipDeviceSynchronize());
    hipfftExecR2C(g->plan_forward, g->d_rho, g->d_F_rho);
    CUDA_ERROR(hipDeviceSynchronize());
    solve_poisson<<<g->gridBlocks, g->gridThreads>>>(g->d_kv, g->d_F_rho, g->d_F_Ex, g->d_F_Ey, g->d_F_Ez, g->N_grid, g->N_grid_all);
    CUDA_ERROR(hipDeviceSynchronize());
    hipfftExecC2R(g->plan_backward, g->d_F_Ex, g->d_Ex);
    hipfftExecC2R(g->plan_backward, g->d_F_Ey, g->d_Ey);
    hipfftExecC2R(g->plan_backward, g->d_F_Ez, g->d_Ez);
    CUDA_ERROR(hipDeviceSynchronize());

    scale_down_after_fft<<<g->gridBlocks, g->gridThreads>>>(g->d_Ex, g->d_Ey, g->d_Ez, g->N_grid, g->N_grid_all);
    CUDA_ERROR(hipDeviceSynchronize());

    reduce_fields<<<(g->N_grid_all + gThreadsAll - 1)/gThreadsAll, gThreadsAll>>>(g->d_rho, g->d_Ex, g->d_Ey, g->d_Ez, g->d_Rrho, g->d_REx, g->d_REy, g->d_REz, g->N_grid_all);
    CUDA_ERROR(hipDeviceSynchronize());
    reduce_fields<<<1, (g->N_grid_all + gThreadsAll - 1)/gThreadsAll>>>(g->d_Rrho, g->d_REx, g->d_REy, g->d_REz, &(g->d_sum_results[0]), &(g->d_sum_results[1]), &(g->d_sum_results[2]), &(g->d_sum_results[3]), (g->N_grid_all + gThreadsAll - 1)/gThreadsAll);
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipMemcpy(g->sum_results, g->d_sum_results, sizeof(float)*4, hipMemcpyDeviceToHost));
    // printf("%f %f %f %f\n", s->moments[0], s->moments[1], s->moments[2], s->moments[3]);
    g->rho_total = g->sum_results[0];
    g->E_total = g->sum_results[1] + g->sum_results[2] + g->sum_results[3];
}


void dump_density_data(Grid *g, char* name){
    CUDA_ERROR(hipMemcpy(g->rho, g->d_rho, sizeof(float)*(g->N_grid_all), hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ex, g->d_Ex, sizeof(float)*(g->N_grid_all), hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ey, g->d_Ey, sizeof(float)*(g->N_grid_all), hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(g->Ez, g->d_Ez, sizeof(float)*(g->N_grid_all), hipMemcpyDeviceToHost));
    FILE *density_data = fopen(name, "w");
    // g->rho_total = 0.0f;
    // g->E_total = 0.0f;
    for (int n = 0; n < g->N_grid_all; n++)
    {
        fprintf(density_data, "%f %.2f %.2f %.2f\n", g->rho[n], g->Ex[n], g->Ey[n], g->Ez[n]);
        // g->rho_total += g->rho[n];
        // g->E_total += g->Ex[n] * g->Ex[n] + g->Ey[n] * g->Ey[n] + g->Ez[n] * g->Ez[n];
    }
    // g->E_total *= 0.5 * EPSILON_ZERO;
}



/*
*   DEBUG SOLVERS
*
*/
void debug_field_solver_uniform(Grid *g){
    float* linear_field_x = new float[g->N_grid_all];
    float* linear_field_y = new float[g->N_grid_all];
    float* linear_field_z = new float[g->N_grid_all];
    for(int i = 0; i<g->N_grid;  i++){
        for(int j = 0; j<g->N_grid;  j++){
            for(int k = 0; k<g->N_grid;  k++){
                int index = i*g->N_grid*g->N_grid + j*g->N_grid + k;
                linear_field_x[index] = 1000;
                linear_field_y[index] = 0;
                linear_field_z[index] = 0;
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
}
void debug_field_solver_sine(Grid *g)
{
    float* linear_field_x = new float[g->N_grid_all];
    float* linear_field_y = new float[g->N_grid_all];
    float* linear_field_z = new float[g->N_grid_all];
    for(int i = 0; i<g->N_grid;  i++){
        for(int j = 0; j<g->N_grid;  j++){
            for(int k = 0; k<g->N_grid;  k++){
                int index = i*g->N_grid*g->N_grid + j*g->N_grid + k;
                linear_field_x[index] = 1000*sin(2*M_PI*((float)k/(float)g->N_grid));
                linear_field_y[index] = 1000*sin(2*M_PI*((float)j/(float)g->N_grid));
                linear_field_z[index] = 1000*sin(2*M_PI*((float)i/(float)g->N_grid));
            }
        }
    }
    hipMemcpy(g->d_Ex, linear_field_x, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ey, linear_field_y, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
    hipMemcpy(g->d_Ez, linear_field_z, sizeof(float)*g->N_grid_all, hipMemcpyHostToDevice);
}

void grid_cleanup(Grid *g)
{
    CUDA_ERROR(hipFree(g->d_rho));
    CUDA_ERROR(hipFree(g->d_Ex));
    CUDA_ERROR(hipFree(g->d_Ey));
    CUDA_ERROR(hipFree(g->d_Ez));
    CUDA_ERROR(hipFree(g->d_F_Ex));
    CUDA_ERROR(hipFree(g->d_F_Ey));
    CUDA_ERROR(hipFree(g->d_F_Ez));
    CUDA_ERROR(hipFree(g->d_F_rho));
}
