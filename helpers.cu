#include "helpers.cuh"

void CUDA_ERROR( hipError_t err){
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s, exiting\n", hipGetErrorString(err));
        exit(-1);
    }
}
